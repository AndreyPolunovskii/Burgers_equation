#include "hip/hip_runtime.h"
#include "include_files.h"
#include "parameters.h"
#include "solver_file.h"
#include "source_funct.cuh"

#include "add_functions.h"

int main_solver(std::fstream &file1,std::fstream &file2,std::fstream &outvar1) {
	
	        scalar T = col_steps * ht;
	        std::cout << " Длина отрезка интегрирования по времени = " << col_steps * ht << std::endl;
	 
	        clock_t time2, time1 = clock();
	
	        int deviceCount;

            //Сколько устройств CUDA установлено на PC.
            hipGetDeviceCount(&deviceCount);
            
            printf("Device count: %d\n\n", deviceCount);
	       
	       // приходится все таки на устройстве работать на чистом Си
	        scalar *dev_x0, *dev_x1; 
	       
	        scalar *x1 ,*x0;
	        	        
	        // размер массива из N чисел
	        ssize_t size = N * sizeof(scalar);
	        
	       // выделяем память на видюхе
	        hipMalloc( (void**)&dev_x0, size );
	        hipMalloc( (void**)&dev_x1, size );

	       
	        // выделяем память в оперативке 	        
	        x0 = (scalar*) malloc( size );
	        x1 = (scalar*) malloc( size );
	        
	       //задаем н. у.
	        for (ssize_t l=1 ; l < N-1; ++l)
                 x0[l] =  ( l < (int)(N - 1)/2) ? 1 : 0;

            // задаем г. у.
            x0[0] = 1;
            x0[N-1] = 0;
            
           // задаем размерности сетки на видеокарте
	       dim3 threads ( BLOCK_SIZE );
           dim3 blocks  ( N / threads.x);
           
           bool flag = 0;
           scalar my_x = 0;
            	       
           // запускаем цикл по времени  
         //---------------------------------------------------------
         for (int k=1 ;k <= col_steps; k++)
        {
		   time2 = clock();
		   
		     // create cuda event handles
            hipEvent_t start, stop;
            float gpuTime = 0.0f;
            hipEventCreate ( &start );
            hipEventCreate ( &stop );
            
             // asynchronously issue work to the GPU (all to stream 0)
            hipEventRecord ( start, 0 );
		   
	       // копируем содержимое переменных из оперативы в переменные на девайсе      
	       hipMemcpy(dev_x0 , x0, size, hipMemcpyHostToDevice );
	        	        
	       ode_euler<<< blocks , threads >>>(dev_x0, dev_x1, mu, hx, a, b, (int)N, ht);
	        
	        // Обратно копируем       
	       hipMemcpy(x1, dev_x1 ,size, hipMemcpyDeviceToHost );
	       
	       //определяем точку завершения работы видеокарты
	       hipEventRecord ( stop, 0 );
           hipEventSynchronize ( stop );
           hipEventElapsedTime ( &gpuTime, start, stop );
	       
	       
	       for (ssize_t i=0 ; i < N; ++i)
	       {
                 x0[i] = x1[i] ;
	           if (x0[i] > 1e+8)
                     flag = 1;
	       }
	        
	        
	        if ( k % (int) (col_steps / col_frame)  == 0)
               {
                  my_x = 0;
                  
                  file1 << k*ht<<" ";
                  for (int j=0 ; j < N; ++j)
                  { 
                     file1 <<  x1[j] << " " ;
                     file2 <<  my_x << " " << x1[j] << std::endl;
                     my_x += hx;
                  }
                  
                  file1 << std::endl;
                  file2 << std::endl;   
                  file2 << std::endl;
                }   
	        
	        
	        
	         if (flag == 1) 
             {
                std::cout << "приближенное решение уходит в бесконечность " << std::endl;
                break;
             }
             
              time1 = clock() - time1;
              time2 = clock() - time2;
             
             if (k % 100 == 0)
            { 
			  printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime );
		      std::cout << k <<"-ый шаг интегрирования" << std::endl;
              std::cout <<"значение " << (int)N/2 <<"-ой расчетной точки: "<< x1[(int)N/2] << " в " << k*ht <<"-ую секунду" << std::endl;
              std::cout << "проинтегрирован отрезок = " << k*ht << std::endl;
              std::cout << "выполнено :" << k*ht*100/T <<"%" << std::endl;
              std::cout << "максимальное значение функции в данный t : " << find_max(x1) << std::endl;


              std::cout << "время, потраченное на одну итерацию :" << ((double) time2)/CLOCKS_PER_SEC << " секунд(ы)" << std::endl;
              std::cout << "общее прошедшее время : " << ((double) time1)/CLOCKS_PER_SEC << " секунд(ы)" << std::endl;
    
              std::cout << "----------------------------" << std::endl;
              std::cout << std::endl;
            }
	        
	        
	     }
	     //---------------------------------------------------------------------
	     
	        
	        // освобождаем память
	        free( x0 ); free( x1 ); 
	        hipFree( dev_x0 );
	        hipFree( dev_x1 );
	        
	        
	        
	        //передаем необходимые переменные в gnuplot

            outvar1 << "col_frame1 = " <<  col_frame << std::endl ;
            outvar1 << "T1 = " <<  T << std::endl ;
            outvar1 << "NUMBER_GR = " <<  10<< std::endl;//col_space_point << endl ;
            outvar1 << "frame_cadr = " << 2 << std::endl;
            outvar1 << "wall_left_gr = " << 0 << std::endl;
            outvar1 << "wall_right_gr = " << L - hx << std::endl;
	                   
            return 0;
}
